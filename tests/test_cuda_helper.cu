#include "mathtools/cuda_helper.cu"
#include "mathtools/matrix_helper.hpp"
#include "io/logger.hpp"
#include "tests/test_utils.hpp"
#include <cassert>


CuHelper cuhelper;

int
NA = 4;

double
sym_matrix_A[] = {
    4, 6, 7, 8,
    6, 9, 2, 1,
    7, 2, 0, 1,
    8, 1, 1, 5 },
diagonal_of_A[] = {
    4, 9, 0, 5,
    6, 2, 1, -1,
    7, 1, -1, -1,
    8, -1, -1, -1},
truth_cblas_dsymv_1[] = { 72.0, 44.0, 22.5, 39.0 },
vector_cblas_dsymv_b_1[] = {4, 5, 6, 7},
matrix_cblas_dsymm_B[] = {
    3, 4, 4, 5,
    1, 8, 3, 5,
    9, 8, 2, 9,
    0, 8, 0, 2},
truth_out_cblas_dsymm [] = {
    104, 67, 34, 57,
    113, 89, 28, 44,
    170, 139, 88, 127,
    64, 74, 18, 18};


int test_cublas_dsymv_1()
{
    LOG::LOGGER.STD("Testing test_cublas_dsymv_1.\n");
    MyCuPtr<double>
        dev_res(NA), dev_sym_matrix_A(NA*NA, sym_matrix_A),
        dev_vector_cblas_dsymv_b_1(NA, vector_cblas_dsymv_b_1);
    double cpu_res[NA];

    cuhelper.dsmyv(
        HIPBLAS_FILL_MODE_UPPER, NA, 0.5, dev_sym_matrix_A.get(), NA,
        dev_vector_cblas_dsymv_b_1.get(), 1, 0, dev_res.get(), 1);
    dev_res.syncDownload(cpu_res, NA);

    if (allClose(truth_cblas_dsymv_1, cpu_res, NA))
        return 0;

    fprintf(stderr, "ERROR test_cublas_dsymv_1.\n");
    printMatrices(truth_cblas_dsymv_1, cpu_res, NA, 1);
    return 1;
}


int test_cublas_dsymm()
{
    LOG::LOGGER.STD("Testing test_cublas_dsymm.\n");
    double result[NA*NA];

    MyCuPtr<double>
        dev_res(NA*NA), dev_sym_matrix_A(NA*NA, sym_matrix_A),
        dev_matrix_cblas_dsymm_B(NA*NA, matrix_cblas_dsymm_B);

    cuhelper.dsymm(
        HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER,
        NA, NA, 1., dev_sym_matrix_A.get(), NA,
        dev_matrix_cblas_dsymm_B.get(), NA,
        0, dev_res.get(), NA);

    dev_res.syncDownload(result, NA*NA);

    if (allClose(truth_out_cblas_dsymm, result, NA*NA))
        return 0;
    fprintf(stderr, "ERROR test_cublas_dsymm.\n");
    printMatrices(truth_out_cblas_dsymm, result, NA, NA);
    return 1;
}


double
matrix_for_SVD_A[] = {
    8.79, 9.93, 9.83, 5.45, 3.16,
    6.11, 6.91, 5.04, -0.27, 7.98,
    -9.15, -7.93, 4.86, 4.85, 3.01,
    9.57, 1.64,8.83, 0.74, 5.8,
    -3.49, 4.02, 9.8, 10.00, 4.27,
     9.84, 0.15, -8.99, -6.02, -5.31},
truth_SVD_A[] = {
    -5.911424e-01, 2.631678e-01, 3.554302e-01, 3.142644e-01, 2.299383e-01,
    -3.975668e-01, 2.437990e-01, -2.223900e-01, -7.534662e-01, -3.635897e-01,
    -3.347897e-02, -6.002726e-01, -4.508393e-01, 2.334497e-01, -3.054757e-01,
    -4.297069e-01, 2.361668e-01,-6.858629e-01, 3.318600e-01, 1.649276e-01,
    -4.697479e-01, -3.508914e-01,  3.874446e-01,  1.587356e-01, -5.182574e-01,
     2.933588e-01,  5.762621e-01, -2.085292e-02,  3.790777e-01, -6.525516e-01};

int
NcolsSVD = 6,
NrowsSVD = 5;

int test_cusolver_SVD()
{
    LOG::LOGGER.STD("Testing test_cusolver_SVD.\n");
    MyCuPtr<double>
        svals(NcolsSVD), dev_svd_matrix(NcolsSVD*NrowsSVD, matrix_for_SVD_A);
    double svd_matrix[NcolsSVD*NrowsSVD];

    cuhelper.svd(dev_svd_matrix.get(), svals.get(), NcolsSVD, NrowsSVD);
    dev_svd_matrix.syncDownload(svd_matrix, NcolsSVD*NrowsSVD);

    if (allClose(truth_SVD_A, svd_matrix, NcolsSVD*NrowsSVD))
        return 0;
    fprintf(stderr, "ERROR test_cusolver_SVD.\n");
    printMatrices(truth_SVD_A, svd_matrix, NrowsSVD, NcolsSVD);
    return 1;
}


int test_cusolver_invert_cholesky()
{
    LOG::LOGGER.STD("Testing test_cusolver_invert_cholesky.\n");
    const int ndim = 496;
    std::vector<double> A, truth_out_inverse;
    int nrows, ncols;

    const std::string
    fname_matrix = std::string(SRCDIR) + "/tests/input/test_symmatrix_cholesky.txt",
    fname_truth  = std::string(SRCDIR) + "/tests/truth/test_inverse_cholesky.txt";

    A = mxhelp::fscanfMatrix(fname_matrix.c_str(), nrows, ncols);
    assert(nrows == ndim);
    assert(ncols == ndim);

    truth_out_inverse = mxhelp::fscanfMatrix(fname_truth.c_str(), nrows, ncols);
    assert(nrows == ndim);
    assert(ncols == ndim);

    MyCuPtr<double> dev_A(ndim*ndim, A.data());

    cuhelper.invert_cholesky(dev_A.get(), ndim);
    dev_A.syncDownload(A.data(), ndim*ndim);
    mxhelp::copyUpperToLower(A.data(), ndim);

    if (allClose(A.data(), truth_out_inverse.data(), ndim*ndim))
        return 0;

    fprintf(stderr, "ERROR test_cusolver_invert_cholesky.\n");
    // printMatrices(truth_out_inverse.data(), A.data(), ndim, ndim);
    return 1;
}


void test_MyCuPtr_init() {
    int NA = 500;
    LOG::LOGGER.STD("Allocating a single MyCuPtr<double>.\n");
    MyCuPtr<double> covariance_matrix;
    assert(covariance_matrix.get() == nullptr);
    covariance_matrix.realloc(NA*NA);
    LOG::LOGGER.STD("Reset a single MyCuPtr<double>.\n");
    covariance_matrix.reset();
    assert(covariance_matrix.get() == nullptr);
}


void test_MyCuPtr_array() {
    int NA = 500;
    LOG::LOGGER.STD("Allocating array of two MyCuPtr<double>.\n");
    MyCuPtr<double> temp_matrix[2];
    assert(temp_matrix[0].get() == nullptr);
    assert(temp_matrix[1].get() == nullptr);
    temp_matrix[0].realloc(NA*NA);
    temp_matrix[1].realloc(NA*NA);
    LOG::LOGGER.STD("Reset array of two MyCuPtr<double>.\n");
    temp_matrix[0].reset();
    temp_matrix[1].reset();
}


void test_MyCuPtr_async() {
    int NA = 500, NB = 5;
    double IN_ARR[] = {1.1, 2.2, 3.3, 4.4, 5.5};

    LOG::LOGGER.STD("Asnyc copy a single MyCuPtr<double>.\n");
    MyCuPtr<double> vec(NB, IN_ARR);
    std::vector<double> cpu_vec(5);
    vec.syncDownload(cpu_vec.data(), 5);
    assert (allClose(cpu_vec.data(), IN_ARR, 5));
}


int main(int argc, char *argv[])
{
    int r=0;

    r += asserter(test_MyCuPtr_init, "test_MyCuPtr_init");
    r += asserter(test_MyCuPtr_array, "test_MyCuPtr_array");
    r += asserter(test_MyCuPtr_async, "test_MyCuPtr_async");

    r += asserter(test_cublas_dsymv_1, "test_cublas_dsymv_1");
    r += asserter(test_cublas_dsymm, "test_cublas_dsymm");
    r += asserter(test_cusolver_SVD, "test_cusolver_SVD");
    r += asserter(
        test_cusolver_invert_cholesky, "test_cusolver_invert_cholesky");

    if (r != 0)
        fprintf(stderr, "ERRORs occured!!!!!\n");
    return r;
}
