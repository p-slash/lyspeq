#ifndef CUDA_HELPER_H
#define CUDA_HELPER_H

#include <stdexcept>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

class MyCuDouble
{
    double *dev_ptr;

    void _alloc(int n) {
        hipError_t stat = hipMalloc((void**) &dev_ptr, n*sizeof(double));
        if (stat != hipSuccess) {
            dev_ptr = nullptr;
            throw std::runtime_error("hipMalloc failed.\n");
        }
    }
public:
    MyCuDouble() { dev_ptr = nullptr; }
    MyCuDouble(int n) { _alloc(n); }
    MyCuDouble(int n, double *cpu_ptr) { 
        _alloc(n); asyncCpy(cpu_ptr, n);
    }
    ~MyCuDouble() { hipFree(dev_ptr); }

    double& operator[](int i) { return dev_ptr[i]; }
    MyCuDouble(const MyCuDouble& udev_ptr) = delete;
    MyCuDouble& operator=(const MyCuDouble& udev_ptr) = delete;

    double* get() const { return dev_ptr; }

    void asyncCpy(double *cpu_ptr, int n, int offset=0) {
        hipMemcpyAsync(dev_ptr + offset, cpu_ptr, n*sizeof(double), hipMemcpyHostToDevice);
    }
    void reset() {
        if (dev_ptr != nullptr) {
            hipFree(dev_ptr);
            dev_ptr = nullptr;
        }
    }

    void realloc(int n, double *cpu_ptr=nullptr) {
        reset(); _alloc(n);
        if (cpu_ptr != nullptr)
            asyncCpy(cpu_ptr, n);
    }
};

class CuHelper
{
    // hipError_t cudaStat;
    hipblasStatus_t blas_stat;
    hipsolverStatus_t solver_stat;

public:
    hipblasHandle_t blas_handle;
    hipsolverHandle_t solver_handle;

    CuHelper() {
        blas_stat = hipblasCreate(&blas_handle);
        if (blas_stat != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("CUBLAS initialization failed.\n");

        solver_stat = hipsolverDnCreate(&solver_handle);
        if (solver_stat != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("CUSOLVER initialization failed.\n");
    };
    ~CuHelper() { hipblasDestroy(blas_handle); hipsolverDnDestroy(solver_handle); };

    // Trace of A.B
    // Assumes A and B square matrices NxN, and at least one to be symmetric.
    // No stride or whatsoever. Continous allocation
    // Uses BLAS dot product.
    double trace_dsymm(const double *A, const double *B, int N) {
        double result;
        blas_stat = hipblasDdot(blas_handle, N*N, A, 1, B, 1, &result);
        if (blas_stat != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("trace_dsymm/hipblasDdot failed.\n");
        return result;
    }

    double trace_ddiagmv(const double *A, const double *B, int N) {
        double result;
        blas_stat = hipblasDdot(blas_handle, N, A, N+1, B, 1, &result);
        if (blas_stat != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("trace_ddiagmv/hipblasDdot failed.\n");
        return result;
    }

    // vT . S . v
    // Assumes S is square symmetric matrix NxN
    double my_cublas_dsymvdot(const double *v, const double *S, double *temp_vector, int N) {
        dsmyv(HIPBLAS_FILL_MODE_UPPER, N, 1., S, N, v, 1, 0, temp_vector, 1);
        double result;
        blas_stat = hipblasDdot(blas_handle, N, v, 1, temp_vector, 1, &result);
        if (blas_stat != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("my_cublas_dsymvdot/hipblasDdot failed.\n");
        return result;
    }

    void dcopy(const double *x, double *y, int N) {
        blas_stat = hipblasDcopy(blas_handle, N, x, 1, y, 1);
        if (blas_stat != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("hipblasDcopy failed.\n");
    }

    void daxpy( double alpha,
                const double *x, double *y,
                int N,
                int incx=1, int incy=1) {
        blas_stat = hipblasDaxpy(blas_handle, N, &alpha, x, incx, y, incy);
        if (blas_stat != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("hipblasDaxpy failed.\n");
    }
    
    void dsymm(hipblasSideMode_t side, hipblasFillMode_t uplo,
               int m, int n, double alpha,
               const double *A, int lda,
               const double *B, int ldb,
               double beta, double *C, int ldc) {
        blas_stat = hipblasDsymm(blas_handle, side, uplo,
            m, n, &alpha,
            A, lda, B, ldb,
            &beta, C, ldc);
        if (blas_stat != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("hipblasDsymm failed.\n");
    }

    void dsmyv( hipblasFillMode_t uplo,
                int n, double alpha,
                const double *A, int lda,
                const double *x, int incx, double beta,
                double *y, int incy) {
        blas_stat = hipblasDsymv(blas_handle, uplo,
            n, &alpha, A, lda, x, incx,
            &beta, y, incy);
        if (blas_stat != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("hipblasDsymv failed.\n");
    }

    // In-place invert by Cholesky factorization
    void invert_cholesky(double *A, int N) {
        int lworkf = 0, lworki = 0; /* size of workspace */
        /* If devInfo = 0, the Cholesky factorization is successful.
        if devInfo = -i, the i-th parameter is wrong (not counting handle).
        if devInfo = i, the leading minor of order i is not positive definite. */
        __device__ int devInfo = -1;

        solver_stat = hipsolverDnDpotrf_bufferSize(
            solver_handle, HIPBLAS_FILL_MODE_UPPER,
            N, A, N, &lworkf);
        if (solver_stat != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDpotrf_bufferSize failed.\n");

        MyCuDouble d_work(lworkf); /* device workspace for getrf */

        solver_stat = hipsolverDnDpotrf(
            solver_handle, HIPBLAS_FILL_MODE_UPPER,
            N, A, N, d_work.get(), lworkf, &devInfo);
        if (solver_stat != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDpotrf failed.\n");
        if (devInfo != 0)
            throw std::runtime_error("Cholesky factorization is not successful.\n");

        solver_stat = hipsolverDnDpotri_bufferSize(
            solver_handle, HIPBLAS_FILL_MODE_UPPER,
            N, A, N, &lworki);
        if (solver_stat != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDpotri_bufferSize failed.\n");

        if (lworki > lworkf)
            d_work.realloc(lworki);

        solver_stat = hipsolverDnDpotri(
            solver_handle, HIPBLAS_FILL_MODE_UPPER,
            N, A, N, d_work.get(), lworki, &devInfo);
        if (solver_stat != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDpotri failed.\n");
        if (devInfo != 0)
            throw std::runtime_error("Cholesky inversion is not successful.\n");
    }

    void svd(double *A, double *svals, int m, int n) {
        int lwork = 0; /* size of workspace */
        __device__ int devInfo = -1;

        solver_stat = hipsolverDnDgesvd_bufferSize(
            solver_handle, m, n, &lwork);
        if (solver_stat != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDgesvd_bufferSize failed.\n");

        MyCuDouble d_work(lwork); /* device workspace */

        solver_stat = hipsolverDnDgesvd(
            solver_handle, 'O', 'N', m, n, A, m, svals,
            nullptr, m, nullptr, n, d_work.get(), lwork,
            nullptr, &devInfo);
        if (solver_stat != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDgesvd failed.\n");
        if (devInfo != 0)
            throw std::runtime_error("SVD is not successful.\n");
    }

};

#endif