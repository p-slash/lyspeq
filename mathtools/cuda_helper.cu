#ifndef CUDA_HELPER_H
#define CUDA_HELPER_H

#include <stdexcept>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>


template<typename T>
class MyCuPtr
{
    T *dev_ptr;

    void _alloc(int n) {
        hipError_t stat = hipMalloc((void**) &dev_ptr, n*sizeof(T));
        if (stat != hipSuccess) {
            dev_ptr = nullptr;
            throw std::runtime_error("hipMalloc failed.");
        }
    }
public:
    MyCuPtr() { dev_ptr = nullptr; }
    MyCuPtr(int n) { _alloc(n); }
    MyCuPtr(int n, T *cpu_ptr, hipStream_t stream=NULL) { 
        _alloc(n); asyncCpy(cpu_ptr, n, 0, stream);
    }
    ~MyCuPtr() { hipFree(dev_ptr); }

    T& operator[](int i) { return dev_ptr[i]; }
    MyCuPtr(const MyCuPtr& udev_ptr) = delete;
    MyCuPtr& operator=(const MyCuPtr& udev_ptr) = delete;

    T* get() const { return dev_ptr; }

    void asyncCpy(T *cpu_ptr, int n, int offset=0, hipStream_t stream=NULL) {
        hipMemcpyAsync(
            dev_ptr + offset, cpu_ptr, n*sizeof(T), hipMemcpyHostToDevice, stream);
    }
    void asyncDwn(T *cpu_ptr, int n, int offset=0, hipStream_t stream=NULL) {
        hipMemcpyAsync(
            cpu_ptr, dev_ptr + offset, sizeof(T) * n, hipMemcpyDeviceToHost, stream);
    }
    void syncDownload(T *cpu_ptr, int n, int offset=0) {
        hipMemcpy(
            cpu_ptr, dev_ptr + offset, n*sizeof(T), hipMemcpyDeviceToHost);
    }
    void reset() {
        if (dev_ptr != nullptr) {
            hipFree(dev_ptr);
            dev_ptr = nullptr;
        }
    }

    void realloc(int n, T *cpu_ptr=nullptr, hipStream_t stream=NULL) {
        reset(); _alloc(n);
        if (cpu_ptr != nullptr)
            asyncCpy(cpu_ptr, n, 0, stream);
    }
};


class MyCuStream {
    hipStream_t stream;
    hipError_t cuda_stat;

    void check_cuda_error(std::string err_msg) {
        if (cuda_stat != hipSuccess) {
            throw std::runtime_error(err_msg);
        }
    }
public:
    MyCuStream() {
        cuda_stat = hipStreamCreateWithFlags(
            &stream, hipStreamNonBlocking);
        check_cuda_error("hipStreamCreateWithFlags: ");
    }
    ~MyCuStream() { hipStreamDestroy(stream); }

    hipStream_t get() const {
        return stream;
    }

    void setCuBLAS(hipblasHandle_t blas_handle) {
        cuda_stat = hipblasSetStream(blas_handle, stream);
        check_cuda_error("hipblasSetStream: ");
    }

    void setCuSOLVER(hipsolverHandle_t solver_handle) {
        cuda_stat = hipsolverSetStream(solver_handle, stream);
        check_cuda_error("hipsolverSetStream: ");
    }

    void sync() {
        cuda_stat = hipStreamSynchronize(stream);
        check_cuda_error("hipStreamSynchronize: ");
    }
}


class CuHelper
{
    // hipError_t cudaStat;
    hipblasStatus_t blas_stat;
    hipsolverStatus_t solver_stat;

    void check_cublas_error(std::string err_msg) {
        if (blas_stat != HIPBLAS_STATUS_SUCCESS) {
            err_msg += std::string(cublasGetStatusString(blas_stat));
            throw std::runtime_error(err_msg);
        }
    }

    void check_cusolver_error(std::string err_msg) {
        if (solver_stat == HIPSOLVER_STATUS_SUCCESS)
            return;
        switch (solver_stat) {
        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            err_msg += "The library was not initialized.";  break;
        case HIPSOLVER_STATUS_INVALID_VALUE:
            err_msg += "Invalid parameters were passed.";   break;
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            err_msg += "An internal operation failed.";     break;
        }

        throw std::runtime_error(err_msg);
    }

public:
    hipblasHandle_t blas_handle;
    hipsolverHandle_t solver_handle;

    CuHelper() {
        blas_stat = hipblasCreate(&blas_handle);
        check_cublas_error("CUBLAS initialization failed: ");

        solver_stat = hipsolverDnCreate(&solver_handle);
        check_cusolver_error("CUSOLVER initialization failed: ");
    };
    ~CuHelper() {
        hipblasDestroy(blas_handle);
        hipsolverDnDestroy(solver_handle);
        hipDeviceReset();
    };

    // Trace of A.B
    // Assumes A and B square matrices NxN, and at least one to be symmetric.
    // No stride or whatsoever. Continous allocation
    // Uses BLAS dot product.
    void trace_dsymm(const double *A, const double *B, int N, double *c_res) {
        blas_stat = hipblasDdot(blas_handle, N*N, A, 1, B, 1, c_res);
        check_cublas_error("trace_dsymm/hipblasDdot: ");
    }

    void trace_ddiagmv(
            const double *A, const double *B, int N, double *c_res) {
        blas_stat = hipblasDdot(blas_handle, N, A, N+1, B, 1, c_res);
        check_cublas_error("trace_ddiagmv/hipblasDdot: ");
    }

    // vT . S . v
    // Assumes S is square symmetric matrix NxN
    void my_cublas_dsymvdot(
            const double *v, const double *S, double *temp_vector, int N,
            double *c_res,
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER) {
        dsmyv(uplo, N, 1., S, N, v, 1, 0, temp_vector, 1);
        blas_stat = hipblasDdot(blas_handle, N, v, 1, temp_vector, 1, c_res);
        check_cublas_error("my_cublas_dsymvdot/hipblasDdot: ");
    }

    void dcopy(const double *x, double *y, int N) {
        blas_stat = hipblasDcopy(blas_handle, N, x, 1, y, 1);
        check_cublas_error("hipblasDcopy: ");
    }

    void daxpy( 
            double alpha,
            const double *x, double *y,
            int N,
            int incx=1, int incy=1) {
        blas_stat = hipblasDaxpy(blas_handle, N, &alpha, x, incx, y, incy);
        check_cublas_error("hipblasDaxpy: ");
    }

    void dsymm(
            hipblasSideMode_t side, hipblasFillMode_t uplo,
            int m, int n, double alpha,
            const double *A, int lda,
            const double *B, int ldb,
            double beta, double *C, int ldc) {
        blas_stat = hipblasDsymm(
            blas_handle, side, uplo,
            m, n, &alpha,
            A, lda, B, ldb,
            &beta, C, ldc);
        check_cublas_error("hipblasDsymm: ");
    }

    void dsmyv(
            hipblasFillMode_t uplo,
            int n, double alpha,
            const double *A, int lda,
            const double *x, int incx, double beta,
            double *y, int incy) {
        blas_stat = hipblasDsymv(
            blas_handle, uplo,
            n, &alpha, A, lda, x, incx,
            &beta, y, incy);
        check_cublas_error("hipblasDsymv: ");
    }

    void potrf(
            double *A, int N,
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER) {
        int lworkf = 0; /* size of workspace */
        /* If devInfo = 0, the Cholesky factorization is successful.
        if devInfo = -i, the i-th parameter is wrong (not counting handle).
        if devInfo = i, the leading minor of order i is not positive definite. */
        MyCuPtr<int> devInfo(1);

        solver_stat = hipsolverDnDpotrf_bufferSize(
            solver_handle, uplo,
            N, A, N, &lworkf);
        check_cusolver_error("hipsolverDnDpotrf_bufferSize: ");

        MyCuPtr<double> d_workf(lworkf); /* device workspace for getrf */

        solver_stat = hipsolverDnDpotrf(
            solver_handle, uplo,
            N, A, N, d_workf.get(), lworkf, devInfo.get());
        check_cusolver_error("hipsolverDnDpotrf: ");
        // if (devInfo != 0)
        //     throw std::runtime_error("Cholesky factorization is not successful.");
    }

    void potri(
            double *A, int N,
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER) {
        int lworki;
        MyCuPtr<int> devInfo(1);
        solver_stat = hipsolverDnDpotri_bufferSize(
            solver_handle, uplo,
            N, A, N, &lworki);
        check_cusolver_error("hipsolverDnDpotri_bufferSize: ");

        MyCuPtr<double> d_worki(lworki);

        solver_stat = hipsolverDnDpotri(
            solver_handle, uplo,
            N, A, N, d_worki.get(), lworki, devInfo.get());
        check_cusolver_error("hipsolverDnDpotri: ");
        // if (devInfo != 0)
        //     throw std::runtime_error("Cholesky inversion is not successful.");
    }

    // In-place invert by Cholesky factorization
    void invert_cholesky(
            double *A, int N,
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER) {
        potrf(A, N, uplo);
        potri(A, N, uplo);
    }

    void svd(double *A, double *svals, int nrows, int ncols) {
        int lwork = 0; /* size of workspace */
        MyCuPtr<int> devInfo(1);
        // __device__ int devInfo = -1;

        solver_stat = hipsolverDnDgesvd_bufferSize(
            solver_handle, nrows, ncols, &lwork);
        check_cusolver_error("hipsolverDnDgesvd_bufferSize: ");

        MyCuPtr<double> d_work(lwork), d_rwork(ncols); /* device workspace */

        solver_stat = hipsolverDnDgesvd(
            solver_handle, 'O', 'N', nrows, ncols, A, nrows, svals,
            nullptr, nrows, nullptr, ncols, d_work.get(), lwork,
            d_rwork.get(), devInfo.get());
        check_cusolver_error("hipsolverDnDgesvd: ");
        // if (devInfo != 0)
        //     throw std::runtime_error("SVD is not successful.");
    }
};

#endif